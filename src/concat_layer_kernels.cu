#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")  
#endif

extern "C" {
#include "concat_layer.h"
#include "blas.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void apply_concat_gpu(float* data_im, float* data_imContext, float* data_imJoint,
        const int height, const int width, const int channel, const int size, const int flip, const int channel_sec) {

    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

    if (index >= size)
        return;

    int col = (index % width);
    index = (index / width);
    int row = (index % height);
    index = (index / height);

    int col_out;
    if(flip)
        col_out = width - col - 1;
    else
        col_out = col;

    for (int c = 0; c < channel; c++){
        data_imJoint[col + width*(row + height*c)]  = data_im[col + width*(row + height*c)];
        //data_imJoint[col_out + width*(row + height*(c+channel))]  = data_imContext[col + width*(row + height*c)];
    }
    for (int c = 0; c < channel_sec; c++){
        //data_imJoint[col + width*(row + height*c)]  = data_im[col + width*(row + height*c)];
        data_imJoint[col_out + width*(row + height*(c+channel))]  = data_imContext[col + width*(row + height*c)];
    }
}

void forward_concat_layer_gpu(concat_layer l, network_state state) {

    float *objInput = state.objNet.layers[state.objNet.n - 1 + l.pointLayer].output_gpu;
    float *contInput = state.contNet.layers[state.contNet.n - 1 + l.pointLayer].output_gpu;
    //fprintf(stderr, "N= %d,  point=%d select=%d\n",state.objNet.n, l.pointLayer,state.objNet.n - 1 + l.pointLayer);
    int total = l.h * l.w;
    int flip = l.flip;
    for(int i = 0; i < l.batch; i++){
        float *a = objInput + i*l.c*l.h*l.w;
        float *b = contInput + i*l.c_sec*l.h*l.w;
        float *c = l.output_gpu + i*l.out_h*l.out_w*l.out_c;
        apply_concat_gpu<<<cuda_gridsize(total), BLOCK>>>(a, b, c, l.h, l.w, l.c, total, flip, l.c_sec);
    }

}
void backward_concat_layer_gpu(concat_layer l, network_state state)
{
}